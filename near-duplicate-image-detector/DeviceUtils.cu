#include "hip/hip_runtime.h"
#include<iostream>
#include<bitset>

#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

#include <hip/hip_runtime.h>
#include"hip/device_functions.h"
#include"hip/hip_runtime.h"

#include"DeviceUtils.cuh"
#include"HostUtils.h"

__device__ unsigned long long compactKernel(bool * arr) {
	unsigned long long result = 0;
	unsigned long long temp;
	for (int i = 0; i < PIXELS; i++) {
		temp = arr[i];
		result |= temp << (PIXELS - i - 1);
	}
	return result;
}

__global__ void compactBatchKernel(bool * d_contiguous, unsigned long long *d_results, int n) {
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= n)
		return;
	d_results[idx] = compactKernel(d_contiguous + (idx * PIXELS));
}

struct hammingFunctor {
	__host__ __device__
		unsigned char operator()(const unsigned long long &a, const unsigned long long &b) const {
		unsigned long long c = a ^ b;
		unsigned char result = 0;

		while (c) {
			result += c & 1;
			c >>= 1;
		}

		return result;
	}
};

std::vector<unsigned long long> batchCompact(thrust::device_vector<bool> d_contiguous) {
	size_t size = d_contiguous.size() / PIXELS;
	thrust::device_vector<unsigned long long> d_results(size);

	// Cast to pointers
	bool* d_contiguousPtr = thrust::raw_pointer_cast(&d_contiguous[0]);
	unsigned long long* d_resultsPtr = thrust::raw_pointer_cast(&d_results[0]);

	// Compact
	size_t blocks = ceil(size / THREADS) + 1;
	compactBatchKernel << <blocks, THREADS >> > (d_contiguousPtr, d_resultsPtr, size);

	// Results
	thrust::host_vector<unsigned long long> h_results = d_results;
	return std::vector<unsigned long long>(h_results.begin(), h_results.end());
}

std::vector<unsigned char> batchHamming(size_t base, thrust::device_vector<unsigned long long> d_hashes) {
	thrust::device_vector<unsigned char> d_distances(d_hashes.size());
	thrust::device_vector<unsigned long long> d_base(d_hashes.size());
	thrust::fill(d_base.begin(), d_base.end(), d_hashes[base]);
	
	// Compute hamming distances
	thrust::transform(d_hashes.begin(), d_hashes.end(), d_base.begin(), d_distances.begin(), hammingFunctor());
	
	thrust::host_vector<unsigned char> h_distances = d_distances;
	return std::vector<unsigned char>(h_distances.begin(), h_distances.end());
}

/*
void test() {
	thrust::device_vector<bool> contiguous(64 * 2);
	for (int i = 0; i < PIXELS; i++)
		contiguous[i] = i % 2;
	for (int i = PIXELS; i < PIXELS * 2; i++)
		contiguous[i] = !(i % 2);

	std::vector<unsigned long long> results = batchCompact(contiguous);
	for (int i = 0; i < results.size(); i++) {
		printf("%d\n", results[i]);
		std::cout << std::bitset<64>(results[i]) << std::endl;
	}
}*/

/*
void test() {
	std::vector<unsigned long long> t(3);
	t[0] = 0x8004022a17938700;	
	t[1] = 0x322454546478e860;
	t[2] = 0xc11a2abad1baa851;

	std::vector<unsigned char> result = batchHamming(1, t);
	for (int i = 0; i < 3; i++) {
		printf("%d ", result[i]);
	}
	printf("\n");
}*/